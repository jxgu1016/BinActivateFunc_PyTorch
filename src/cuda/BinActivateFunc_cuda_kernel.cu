#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

// CUDA: grid stride looping
#define CUDA_KERNEL_LOOP(i, n) \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

namespace {
template <typename scalar_t>
__global__ void BinActivateFunc_cuda_backward_kernel(
    const int nthreads,
    const scalar_t* __restrict__ input_data,
    scalar_t* __restrict__ gradInput_data) 
{
    CUDA_KERNEL_LOOP(n, nthreads) {
        if (*(input_data + n) > 1 || *(input_data + n) < -1) {
            *(gradInput_data + n) = 0;
        }
    }
}
} // namespace

int BinActivateFunc_cuda_backward(
    at::Tensor input,
    at::Tensor gradInput) 
{
    const int nthreads = input.numel();
    const int CUDA_NUM_THREADS = 1024;
    const int nblocks = (nthreads + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;

    AT_DISPATCH_FLOATING_TYPES(input.type(), "BinActivateFunc_cuda_backward", ([&] {
        BinActivateFunc_cuda_backward_kernel<scalar_t><<<nblocks, CUDA_NUM_THREADS>>>(
            nthreads,
            input.data<scalar_t>(),
            gradInput.data<scalar_t>());
    }));
    return 1;
}
